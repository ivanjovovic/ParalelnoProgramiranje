
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Matrix size
#define N 2500

// Thread block size
#define BLOCK_SIZE 32

__global__ void MatrixMulKernel(float *a, float *b, float *c)
{
    // 2D Thread ID
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    // Output element
    float Pvalue = 0;

    // Loop over all A and B elements
    for (int k = 0; k < N; ++k)
    {
        // Multiply-add
        Pvalue += a[tx * N + k] * b[k * N + ty];

    }

    // Save result
    c[ty * N + tx] = Pvalue;
}

int main()
{
    // Allocate host memory
    float *h_a = (float*)malloc(N * N * sizeof(float));
    float *h_b = (float*)malloc(N * N * sizeof(float));
    float *h_c = (float*)malloc(N * N * sizeof(float));

    // Initialize host memory
    for (int i = 0; i < N * N; ++i)
    {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
        h_c[i] = 0;
    }

    // Allocate device memory
    float *d_a;
    float *d_b;
    float *d_c;
    hipMalloc((void**)&d_a, N * N * sizeof(float));
    hipMalloc((void**)&d_b, N * N * sizeof(float));
    hipMalloc((void**)&d_c, N * N * sizeof(float));

    // Copy host memory to device
    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Setup execution configuration
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Start timer
    clock_t start = clock();

    // Launch kernel on the device
    MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Stop timer
    clock_t end = clock();

    // Print time
    printf("GPU computation time: %f s\n", (double)(end - start) / CLOCKS_PER_SEC);
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Start timer
    start = clock();

    // CPU matrix multiplication
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            for (int k = 0; k < N; ++k)
                h_c[i * N + j] += h_a[i * N + k] * h_b[k * N + j];

    // Stop timer
    end = clock();

    // Print time
    printf("CPU computation time: %f s\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Free host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
// Shutdown CUDA
    hipDeviceReset();

    return 0;
}
